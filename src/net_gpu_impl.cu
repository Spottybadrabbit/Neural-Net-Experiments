#include "hip/hip_runtime.h"
#include "net_gpu_impl.hpp"
#include <host_config.h>
#include <iostream>
#include <stdio.h>
#include "functions.hpp"

inline size_t matrix_size(const Raw_Matrix & m) {
  return sizeof(float) * m.n_rows * m.n_cols;
}

Raw_Matrix * matrix_to_gpu(Raw_Matrix & source) {
  Raw_Matrix h_mat;
  hipMalloc((void **) &(h_mat.data), matrix_size(source));
  hipMemcpy(h_mat.data, source.data, matrix_size(source), hipMemcpyHostToDevice);
  h_mat.n_rows = source.n_rows;
  h_mat.n_cols = source.n_cols;
  Raw_Matrix *d_ma;
  hipMalloc((void **) &d_ma, sizeof(Raw_Matrix));
  hipMemcpy(d_ma, &h_mat, sizeof(Raw_Matrix), hipMemcpyHostToDevice);
  return d_ma;
}

Raw_Matrix matrix_to_cpu(Raw_Matrix * d_mat) {
  Raw_Matrix h_mat;
  hipMemcpy(&h_mat, d_mat, sizeof(Raw_Matrix), hipMemcpyDeviceToHost);
  float * d_data = h_mat.data;
  h_mat.data = new float[h_mat.n_rows * h_mat.n_cols];
  hipMemcpy(h_mat.data, d_data, matrix_size(h_mat), hipMemcpyDeviceToHost);
  return h_mat;
}

void copy_matrix_with_gpu_ptr(Raw_Matrix & dst, Raw_Matrix & src) {
  hipMalloc((void **) &(dst.data), matrix_size(src));
  hipMemcpy(dst.data, src.data, matrix_size(src), hipMemcpyHostToDevice);
}

template<typename activation, typename error>
Raw_FeedForward_Network<activation, error> * network_to_gpu(Raw_FeedForward_Network<activation, error> & network) {
  Raw_FeedForward_Network<activation, error> h_network = network;

  copy_matrix_with_gpu_ptr(h_network.weights_inputToHidden, network.weights_inputToHidden);

  copy_matrix_with_gpu_ptr(h_network.weights_hiddenToOutput, network.weights_hiddenToOutput);

  copy_matrix_with_gpu_ptr(h_network.activation_input, network.activation_input);
  copy_matrix_with_gpu_ptr(h_network.activation_hidden, network.activation_hidden);
  copy_matrix_with_gpu_ptr(h_network.activation_output, network.activation_output);

  Raw_FeedForward_Network<activation, error> * d_network;
  hipMalloc((void **)&d_network, sizeof(Raw_FeedForward_Network<activation, error>));
  hipMemcpy(d_network, &h_network, sizeof(Raw_FeedForward_Network<activation, error>), hipMemcpyHostToDevice);

  return d_network;
}

template<typename activation, typename error>
void network_to_cpu(Raw_FeedForward_Network<activation, error> * d_network,
    Raw_FeedForward_Network<activation, error> & h_network) {
  Raw_FeedForward_Network<activation, error> orig_address_network = h_network;
  hipMemcpy(&h_network, d_network, sizeof(Raw_FeedForward_Network<activation, error>), hipMemcpyDeviceToHost);
  Raw_FeedForward_Network<activation, error> gpu_address = h_network;

  h_network.weights_inputToHidden.data = orig_address_network.weights_inputToHidden.data;
  h_network.weights_hiddenToOutput.data = orig_address_network.weights_hiddenToOutput.data;
  h_network.activation_input.data = orig_address_network.activation_input.data;
  h_network.activation_hidden.data = orig_address_network.activation_hidden.data;
  h_network.activation_output.data = orig_address_network.activation_output.data;

  hipMemcpy(h_network.weights_inputToHidden.data, gpu_address.weights_inputToHidden.data,
      matrix_size(h_network.weights_inputToHidden), hipMemcpyDeviceToHost);

  hipMemcpy(h_network.weights_hiddenToOutput.data, gpu_address.weights_hiddenToOutput.data,
      matrix_size(h_network.weights_hiddenToOutput), hipMemcpyDeviceToHost);

  hipMemcpy(h_network.activation_input.data, gpu_address.activation_input.data,
      matrix_size(h_network.activation_input), hipMemcpyDeviceToHost);

  hipMemcpy(h_network.activation_hidden.data, gpu_address.activation_hidden.data,
      matrix_size(h_network.activation_hidden), hipMemcpyDeviceToHost);

  hipMemcpy(h_network.activation_output.data, gpu_address.activation_output.data,
      matrix_size(h_network.activation_output), hipMemcpyDeviceToHost);
}


const int block_size = 64;

template<typename activation, typename error>
__global__ void kernel_calculate_hidden_activations(Raw_FeedForward_Network<activation, error> * d_network) {
  
  Raw_Matrix & prior_activation = d_network->activation_input;
  Raw_Matrix & post_activation = d_network->activation_hidden;
  Raw_Matrix & weights = d_network->weights_inputToHidden;
  
  int index = blockIdx.x * block_size + threadIdx.x;
  if (index >= post_activation.n_rows * prior_activation.n_cols) {
    return;
  }
  int on_activation= index / prior_activation.n_rows;
  int on_trial= index % post_activation.n_rows;
  post_activation.at(on_activation, on_trial) = 0;
  for (int i=0; i < weights.n_rows; ++i) {
    post_activation.at(on_trial, on_activation) += weights.at(i, on_activation) * prior_activation.at(on_trial, i);
  }
  post_activation.at(on_trial, on_activation) = activation::activation(post_activation.at(on_trial, on_activation));
}

template<typename activation, typename error>
__global__ void kernel_calculate_output_activations(Raw_FeedForward_Network<activation, error> * d_network) {

  Raw_Matrix & prior_activation = d_network->activation_hidden;
  Raw_Matrix & post_activation = d_network->activation_output;
  Raw_Matrix & weights = d_network->weights_hiddenToOutput;
  int index = blockIdx.x * block_size + threadIdx.x;
  if (index >= post_activation.n_rows * prior_activation.n_cols) {
    return;
  }
  int on_activation= index / prior_activation.n_rows;
  int on_trial= index % post_activation.n_rows;
  post_activation.at(on_activation, on_trial) = 0;
  for (int i=0; i < weights.n_rows; ++i) {
    post_activation.at(on_trial, on_activation) += weights.at(i, on_activation) * prior_activation.at(on_trial, i);
  }
  post_activation.at(on_trial, on_activation) = activation::activation(post_activation.at(on_trial, on_activation));

}

template<typename activation, typename error>
__global__ void kernel_set_input_activations(Raw_FeedForward_Network<activation, error> * d_network, Raw_Matrix * input) {

  int index = blockIdx.x * block_size + threadIdx.x;
  int num_trials = input->n_rows;
  int feature_size = input->n_cols;
  if (index >= feature_size * num_trials) {
    return;
  }
  int on_trial = index / input->n_cols;
  int on_activation = index % input->n_cols;

  d_network->activation_input.at(on_trial, on_activation) = input->at(on_trial, on_activation);
}

template<typename activation, typename error>
void calculate_activation(int num_trials, int input_size, int hidden_size, int output_size, Raw_FeedForward_Network<activation, error> * d_network, Raw_Matrix * d_input)
{
  kernel_set_input_activations<<<1 + num_trials * input_size / block_size, block_size>>>(d_network, d_input);
  kernel_calculate_hidden_activations<<<1 + num_trials * hidden_size / block_size, block_size>>>(d_network);
  kernel_calculate_output_activations<<<1 + num_trials * output_size / block_size, block_size>>>(d_network);
}

template Raw_FeedForward_Network<Logistic, Squared_Error> * network_to_gpu(Raw_FeedForward_Network<Logistic, Squared_Error> & source);
template void network_to_cpu(Raw_FeedForward_Network<Logistic, Squared_Error> * d_network,
    Raw_FeedForward_Network<Logistic, Squared_Error> & h_network);

template void calculate_activation(int num_trials, int input_size, int hidden_size, int output_size, Raw_FeedForward_Network<Logistic, Squared_Error> * d_network, Raw_Matrix * d_input);

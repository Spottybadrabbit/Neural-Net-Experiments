#include "hip/hip_runtime.h"
#include "net_gpu_impl.hpp"
#include <host_config.h>
#include <iostream>
#include <stdio.h>

inline size_t matrix_size(const Raw_Matrix & m) {
  return sizeof(float) * m.n_rows * m.n_cols;
}

Raw_Matrix * matrix_to_gpu(Raw_Matrix & source) {
  Raw_Matrix h_mat;
  hipMalloc((void **) &(h_mat.data), matrix_size(source));
  hipMemcpy(h_mat.data, source.data, matrix_size(source), hipMemcpyHostToDevice);
  h_mat.n_rows = source.n_rows;
  h_mat.n_cols = source.n_cols;
  Raw_Matrix *d_ma;
  hipMalloc((void **) &d_ma, sizeof(Raw_Matrix));
  hipMemcpy(d_ma, &h_mat, sizeof(Raw_Matrix), hipMemcpyHostToDevice);
  return d_ma;
}

Raw_Matrix matrix_to_cpu(Raw_Matrix * d_mat) {
  Raw_Matrix h_mat;
  hipMemcpy(&h_mat, d_mat, sizeof(Raw_Matrix), hipMemcpyDeviceToHost);
  float * d_data = h_mat.data;
  h_mat.data = new float[h_mat.n_rows * h_mat.n_cols];
  hipMemcpy(h_mat.data, d_data, matrix_size(h_mat), hipMemcpyDeviceToHost);
  return h_mat;
}

void copy_matrix_with_gpu_ptr(Raw_Matrix & dst, Raw_Matrix & src) {
  hipMalloc((void **) &(dst.data), matrix_size(src));
  hipMemcpy(dst.data, src.data, matrix_size(src), hipMemcpyHostToDevice);
}

Raw_FeedForward_Network * network_to_gpu(Raw_FeedForward_Network & network) {
  Raw_FeedForward_Network h_network = network;
  
  //copy_matrix_with_gpu_ptr(h_network.weights_inputToHidden, network.weights_inputToHidden);
  hipMalloc((void **) &(h_network.weights_inputToHidden.data),
      matrix_size(h_network.weights_inputToHidden));

  hipMemcpy(h_network.weights_inputToHidden.data, h_network.weights_inputToHidden.data,
      matrix_size(h_network.weights_inputToHidden), hipMemcpyHostToDevice);

  copy_matrix_with_gpu_ptr(h_network.weights_hiddenToOutput, network.weights_hiddenToOutput);
  
  copy_matrix_with_gpu_ptr(h_network.activation_input, network.activation_input);
  copy_matrix_with_gpu_ptr(h_network.activation_hidden, network.activation_hidden);
  copy_matrix_with_gpu_ptr(h_network.activation_output, network.activation_output);

  Raw_FeedForward_Network * d_network;
  hipMalloc((void **)&d_network, sizeof(Raw_FeedForward_Network));
  hipMemcpy(d_network, &h_network, sizeof(Raw_FeedForward_Network), hipMemcpyHostToDevice);

  return d_network;
}

void network_to_cpu(Raw_FeedForward_Network * d_network,
    Raw_FeedForward_Network & h_network) {
  Raw_FeedForward_Network orig_address_network = h_network;
  hipMemcpy(&h_network, d_network, sizeof(Raw_FeedForward_Network), hipMemcpyDeviceToHost);
  Raw_FeedForward_Network gpu_address = h_network;

  h_network.weights_inputToHidden.data = orig_address_network.weights_inputToHidden.data;
  h_network.weights_hiddenToOutput.data = orig_address_network.weights_hiddenToOutput.data;
  h_network.activation_input.data = orig_address_network.activation_input.data;
  h_network.activation_hidden.data = orig_address_network.activation_hidden.data;
  h_network.activation_output.data = orig_address_network.activation_output.data;

  hipMemcpy(h_network.weights_inputToHidden.data, gpu_address.weights_inputToHidden.data,
      matrix_size(h_network.weights_inputToHidden), hipMemcpyDeviceToHost);

  hipMemcpy(h_network.weights_hiddenToOutput.data, gpu_address.weights_hiddenToOutput.data,
      matrix_size(h_network.weights_hiddenToOutput), hipMemcpyDeviceToHost);

  hipMemcpy(h_network.activation_input.data, gpu_address.activation_input.data,
      matrix_size(h_network.activation_input), hipMemcpyDeviceToHost);

  hipMemcpy(h_network.activation_hidden.data, gpu_address.activation_hidden.data,
      matrix_size(h_network.activation_hidden), hipMemcpyDeviceToHost);

  hipMemcpy(h_network.activation_output.data, gpu_address.activation_output.data,
      matrix_size(h_network.activation_output), hipMemcpyDeviceToHost);
}


__global__ void kernel_raw_predict(Raw_FeedForward_Network * d_network, Raw_Matrix * input, Raw_Matrix * output) {
  printf ("Hello from inside kernel %d\n", blockIdx.x);
}

Raw_Matrix raw_predict_gpu(Raw_FeedForward_Network & network, Raw_Matrix & input) {
  /*
  Raw_FeedForward_Network * d_network = to_gpu(network);
  Raw_Matrix * d_output;
  hipMalloc((void *jjj*) &d_output, sizeof(Raw_Matrix));
  Raw_Matrix * d_input  = malloc_Matrix(input);
  hipMalloc((void **) &d_output->data, sizeof(float) * network.output_size * input.n_rows);

  kernel_raw_predict<<<input.n_rows, 1>>>(d_network, d_input, d_output);
  */
  Raw_Matrix r;
  return r;
}

void raw_train_batch_gpu(Raw_FeedForward_Network & network, Raw_Matrix & inputs,
    Raw_Matrix & targets, float learning_rate, int batch_size) {

}
